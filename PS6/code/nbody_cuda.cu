#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>


#define dT 0.2f
#define G 0.6f
#define BLOCK_SIZE 64

// Global variables
int num_planets;
int num_timesteps;

// Host arrays
float2* velocities;
float4* planets;

// Device arrays 
float2* velocities_d;
float4* planets_d;


// Parse command line arguments
void parse_args(int argc, char** argv){
    if(argc != 2){
        printf("Useage: nbody num_timesteps\n");
        exit(-1);
    }
    
    num_timesteps = strtol(argv[1], 0, 10);
}

// Reads planets from planets.txt
void read_planets(){

    FILE* file = fopen("planets256.txt", "r");
    if(file == NULL){
        printf("'planets.txt' not found. Exiting\n");
        exit(-1);
    }

    char line[200];
    fgets(line, 200, file);
    sscanf(line, "%d", &num_planets);

    planets = (float4*)malloc(sizeof(float4)*num_planets);
    velocities = (float2*)malloc(sizeof(float2)*num_planets);

    for(int p = 0; p < num_planets; p++){
        fgets(line, 200, file);
        sscanf(line, "%f %f %f %f %f",
                &planets[p].x,
                &planets[p].y,
                &velocities[p].x,
                &velocities[p].y,
                &planets[p].z);
    }

    fclose(file);
}

// Writes planets to file
void write_planets(int timestep){
    char name[20];
    int n = sprintf(name, "planets_out.txt");

    FILE* file = fopen(name, "wr+");

    for(int p = 0; p < num_planets; p++){
        fprintf(file, "%f %f %f %f %f\n",
                planets[p].x,
                planets[p].y,
                velocities[p].x,
                velocities[p].y,
                planets[p].z);
    }

    fclose(file);
}

// TODO 7. Calculate the change in velocity for p, caused by the interaction with q
__device__ float2 calculate_velocity_change_planet(float4 p, float4 q){

}

// TODO 5. Calculate the change in velocity for my_planet, caused by the interactions with a block of planets
__device__ float2 calculate_velocity_change_block(float4 my_planet, float4* shared_planets){

}

// TODO 4. Update the velocities by calculating the planet interactions
__global__ void update_velocities(float4* planets, float2* velocities, int num_planets){

}

// TODO 7. Update the positions of the planets using the new velocities
__global__ void update_positions(float4* planets, float2* velocities, int num_planets){

}


int main(int argc, char** argv){

    parse_args(argc, argv);
    read_planets();

    // TODO 1. Allocate device memory, and transfer data to device 
    hipMalloc(&velocities_d, sizeof(float2)*num_planets);
    hipMalloc(&planets_d, sizeof(float4)*num_planets);

    hipMemcpy(velocities_d, velocities, sizeof(float2)*num_planets, hipMemcpyHostToDevice);
    hipMemcpy(planets_d, planets, sizeof(float4)*num_planets, hipMemcpyHostToDevice);

    // Calculating the number of blocks
    int num_blocks = num_planets/BLOCK_SIZE + ((num_planets%BLOCK_SIZE == 0) ? 0 : 1);

    // Main loop
    for(int t = 0; t < num_timesteps; t++){
        // TODO 2. Call kernels
    }

    // TODO 3. Transfer data back to host
    hipMemcpy(velocities, velocities_d, sizeof(float2)*num_planets, hipMemcpyDeviceToHost);
    hipMemcpy(planets, planets_d, sizeof(float4)*num_planets, hipMemcpyDeviceToHost);

    // Output
    write_planets(num_timesteps);
}
