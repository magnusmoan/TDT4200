#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define dT 0.2f
#define G 0.6f
#define BLOCK_SIZE 64

// Global variables
int num_planets;
int num_timesteps;

// Host arrays
float2* velocities;
float4* planets;

// Device arrays 
float2* velocities_d;
float4* planets_d;


// Parse command line arguments
void parse_args(int argc, char** argv){
    if(argc != 2){
        printf("Useage: nbody num_timesteps\n");
        exit(-1);
    }
    
    num_timesteps = strtol(argv[1], 0, 10);
}

// Reads planets from planets.txt
void read_planets(){

    FILE* file = fopen("planets.txt", "r");
    if(file == NULL){
        printf("'planets.txt' not found. Exiting\n");
        exit(-1);
    }

    char line[200];
    fgets(line, 200, file);
    sscanf(line, "%d", &num_planets);

    planets = (float4*)malloc(sizeof(float4)*num_planets);
    velocities = (float2*)malloc(sizeof(float2)*num_planets);

    for(int p = 0; p < num_planets; p++){
        fgets(line, 200, file);
        sscanf(line, "%f %f %f %f %f",
                &planets[p].x,
                &planets[p].y,
                &velocities[p].x,
                &velocities[p].y,
                &planets[p].z);
    }

    fclose(file);
}

// Writes planets to file
void write_planets(int timestep){
    char name[20];
    int n = sprintf(name, "planets_out.txt");

    FILE* file = fopen(name, "wr+");

    for(int p = 0; p < num_planets; p++){
        fprintf(file, "%f %f %f %f %f\n",
                planets[p].x,
                planets[p].y,
                velocities[p].x,
                velocities[p].y,
                planets[p].z);
    }

    fclose(file);
}

// TODO 6. Calculate the change in velocity for p, caused by the interaction with q
__device__ float2 calculate_velocity_change_planet(float4 p, float4 q){
    float2 vChange;
    float2 acc;

    acc.x = q.x - p.x;
    acc.y = q.y - p.y;

    float dist = sqrt(acc.x*acc.x + acc.y*acc.y);
    float cubed = dist*dist*dist;

    vChange.x = dT*G*q.mass/cubed * dist.x;
    vChange.y = dT*G*q.mass/cubed * dist.y;

    return vChange;

}

// TODO 5. Calculate the change in velocity for my_planet, caused by the interactions with a block of planets
__device__ float2 calculate_velocity_change_block(float4 my_planet, float4* shared_planets){
    float2 v = float2(0.0, 0.0);
    int i;

    for(i = 0; i < blockDim.x; ++i) {
        float2 change = calculate_velocity_change_planet(my_planet, shared_planets[i]);
	v.x += change.x;
	v.y += change.y;
    }

    return v;
}

// TODO 4. Update the velocities by calculating the planet interactions
__global__ void update_velocities(float4* planets, float2* velocities, int num_planets){
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    float4 planet = planets[id];

    __shared__ float4 shared_planets[BLOCK_SIZE];
    int i;
    for(i = 0; i < num_planets; i+=blockDim.x) {
	shared_planets[threadIdx.x] = planets[i + threadIdx.x];
	__syncthreads();

	float2 v = calculate_velocity_change_block(planet, shared_planets);
	velocities[id].x += v.x;
	velocities[id].y += v.y;
	__syncthreads();
    }
}

// TODO 7. Update the positions of the planets using the new velocities
__global__ void update_positions(float4* planets, float2* velocities, int num_planets){
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    planets[id].x += velocities[id].x * dT;
    planets[id].y += velocities[id].y * dT;

}


int main(int argc, char** argv){

    parse_args(argc, argv);
    read_planets();

    // TODO 1. Allocate device memory, and transfer data to device 
    hipMalloc(&velocities_d, sizeof(float2)*num_planets);
    hipMalloc(&planets_d, sizeof(float4)*num_planets);

    hipMemcpy(velocities_d, velocities, sizeof(float2)*num_planets, hipMemcpyHostToDevice);
    hipMemcpy(planets_d, planets, sizeof(float4)*num_planets, hipMemcpyHostToDevice);

    // Calculating the number of blocks
    int num_blocks = num_planets/BLOCK_SIZE + ((num_planets%BLOCK_SIZE == 0) ? 0 : 1);

    // Main loop
    for(int t = 0; t < num_timesteps; t++){
        // TODO 2. Call kernels
	update_velocities<<<num_blocks, BLOCK_SIZE>>>(planets_d, velocities_d, num_planets);
	update_positions<<<num_blocks, BLOCK_SIZE>>>(planets_d, velocities_d, num_planets);

    }

    // TODO 3. Transfer data back to host
    hipMemcpy(velocities, velocities_d, sizeof(float2)*num_planets, hipMemcpyDeviceToHost);
    hipMemcpy(planets, planets_d, sizeof(float4)*num_planets, hipMemcpyDeviceToHost);

    // Output
    write_planets(num_timesteps);
}
